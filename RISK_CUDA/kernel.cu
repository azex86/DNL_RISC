#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <chrono>
#include <string>
using namespace std;
void help();
#define SET_RESULT_WIN(value) (value<<32)
#define SET_RESULT_MEAN_RESTE(value) (value)
#define SET_RETURN_ECHANTILLONNAGE(win,reste) (SET_RESULT_WIN((__int64)win) | SET_RESULT_MEAN_RESTE((__int64)reste))
__device__ char dice(hiprandState_t& state)
{
    return 1 + (char)(hiprand_uniform(&state) * (6 - 1 + 1));
}

__device__ inline void sort(short* data, short size)
{
    if (size == 1)
        return;
    else if (size == 2)
    {
        
        if (data[0] < data[1])
        {
            short temp;
            temp = data[0];
            data[0] = data[1];
            data[1] = temp;
        }
        return;
    }
    else
    {
        short temp;
#define A data[0]
#define B data[1]
#define C data[2]
        
        if (A>B)
        {
            if (C > B)
            {
                temp = B;
                B = C;
                C = temp;
            }
            else {
                temp = A;
                A = C;
                C = B;
                B = temp;
            }
        }
        else
        {
            if (B > C)
            {
                if (A > C)
                {
                    temp = A;
                    A = B;
                    B = temp;
                }
                else
                {
                    temp = A;
                    A = B;
                    B = C;
                    C = temp;
                }
            }
            else
            {
                temp = A;
                A = C;
                C = temp;
            }
        }
    }
}

__device__ int simulateFight(int defense, int attack, hiprandState_t& state)
{
    short die_defend[3];
    short die_attack[3];
    while (defense > 0 && attack > 0)
    {
        if (attack > 2)
        {
            die_attack[0]=  dice(state);
            die_attack[1] = dice(state);
            die_attack[2] = dice(state);
        }
        else if (attack == 2)
        {
            die_attack[0] = dice(state);
            die_attack[1] = dice(state);
        }
        else
        {
            die_attack[0] = dice(state);
        }
        sort(die_attack, attack);
        

        if (defense > 1)
        {
            die_defend[0] = dice(state);
            die_defend[1] = dice(state);
        }
        else
        {
            die_defend[0] = dice(state);
        }
        sort(die_defend, defense);

        int _min = min(attack, defense);
        if (die_attack[0] > die_defend[0])
        {
            defense--;
        }
        else
            attack--;

        if (_min > 1)
        {
            if (die_attack[1] > die_defend[1])
            {
                defense--;
            }
            else
                attack--;
        }
    }

    return  (attack > 0) ? (attack) : (-defense);
}


__global__ void makeRandom(int *data, unsigned int size,unsigned int n_thread,int seed,int def,int att)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int id = threadIdx.x;
    int start = size/n_thread * id + min(size%n_thread,id) ;
    int end = size /n_thread*(id+1) + min(size % n_thread, id + 1);
    hiprandState_t state;

    hiprand_init(seed, tid, 0, &state); // Initialisation du générateur de nombres aléatoires
    for (int i = start; i < end; i++)
    {
        
        data[i] = simulateFight(def,att,state);
    }
    
}

int main(int argc,char **argv)
{
    int n_thread = 100,n_block = 1;
    int n_defence = 3, n_attack = 3;
    int n_echantillon = 1000;
    bool verbose = true;
    for (int i = 0; i < argc; i++)
    {
        if (strcmp(argv[i], "--def") == 0)
        {
            n_defence = std::stoi(argv[++i]);
        }
        else
            if (strcmp(argv[i], "--att") == 0)
            {
                n_attack = std::stoi(argv[++i]);
            }
            else
                if (strcmp(argv[i], "--thread") == 0)
                {
                    n_thread = std::stoi(argv[++i]);
                }
                else
                    if (strcmp(argv[i], "--N") == 0)
                    {
                        n_echantillon = std::stoi(argv[++i]);
                    }
                    else
                        if (strcmp(argv[i], "-h") == 0 || strcmp(argv[i], "--help") == 0)
                        {
                            help();
                            return 0;
                        }
                        else
                            if (strcmp(argv[i], "-q") == 0 || strcmp(argv[i], "--quiet") == 0)
                            {
                                verbose = false;
                            }
    }


    auto start = std::chrono::high_resolution_clock::now();


    int* cpu_data = new int[n_echantillon];
    int* gpu_data = nullptr;
    hipMalloc(&gpu_data, n_echantillon * sizeof(int));
    unsigned int seed = (unsigned int)time(NULL); // Utiliser le temps actuel comme graine

    makeRandom << <n_block, n_thread >> > (gpu_data, n_echantillon, n_thread,seed, n_defence,n_attack);
    hipDeviceSynchronize();
    hipMemcpy(cpu_data, gpu_data, n_echantillon *sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);


    int n_win = 0;
    int n_remaining = 0;
  
    for (int i = 0; i < n_echantillon; i++)
    {
        n_remaining += cpu_data[i];
        if (cpu_data[i] > 0)
            n_win++;
    }
    
    if (verbose)
    {
        double win_rate = (double)n_win / n_echantillon;
        double remaining_mean = (double)n_remaining / n_echantillon;

        cout << "Result : " << n_win << " victoires soit a rate of : " << (double)(n_win) / n_echantillon << " with a mean of " << (double)n_remaining / n_echantillon << " soldier alive at the end" << std::endl;
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> temps_generation = end - start;
        std::cout << "Temps de generation : " << temps_generation.count() << " secondes" << std::endl;
    }
    cout.write((char*)&n_win, 4);
    cout.write((char*)&n_remaining, 4);
    return 0;
}


void help()
{
    cout << "Programme de simulation de calcul pour le jeu RISK (rien à voir avec RISC) from aze\n\
            arguments disponible : \n\
                -h --help : affiche ce message d'aide\n\
                --N : defini le nombre d'execution du calul avant de rencoyer la somme\n\
                --def : defini le nombre de soldat defenseur\n\
                --att : defini le nombre de soldat attaquants\n\
                --thread : defini le nombre de thread a utiliser pour les calculs\n\
                -q --quiet : desactive l'usage de la sortie standard pour un usage autre que le resultat\n\
\n\n            La valeur de retour designe ici les 8 derniers octets ecrit sur la sortie standard\n\
                La valeur de retour est encode comme ceci : \n\
                le signe indique le camp des soldats restant + pour les attauqants - pour les defenseurs\n\
                les 4 premiers octets indique le nombre de victoire des attaquants sur les N echantillons\n\
                les 4 dernniers octets indiquer le nombre de soldat restant à la fin de la bataille\n\
                il est attendu que ces octets soit pris sur la valeur absolue du code de retour.\n";



}